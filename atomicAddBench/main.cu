
#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"

#include "stdlib.h"
#include "stdio.h"
#include "time.h"
#include <typeinfo>

#define VERBOSE 0
#define INTEGER_SCALE_FACTOR 100

#define DEFAULT_NUM_ITERATIONS 1
#define DEFAULT_NUM_ELEMENTS 128
#define DEFAULT_SEED 0

#ifdef WIN32
#define EXE_NAME "atomicAddBench.exe"
#else
#define EXE_NAME "atomicAddBench"
#endif

#define MIN_ARGS 1
#define MAX_ARGS 4
#define ARG_ITERATIONS 1
#define ARG_ELEMENTS 2
#define ARG_SEED 3

static void HandleError(const char *file, int line, hipError_t status = hipGetLastError()) {
	if (status != hipSuccess || (status = hipGetLastError()) != hipSuccess)
	{
		if (status == hipErrorUnknown)
		{
			printf("%s(%i) An Unknown CUDA Error Occurred :(\n", file, line);
			exit(1);
		}
		printf("%s(%i) CUDA Error Occurred;\n%s\n", file, line, hipGetErrorString(status));
		exit(1);
	}
}

#define CUDA_CALL( err ) (HandleError(__FILE__, __LINE__ , err))
#define CUDA_CHECK() (HandleError(__FILE__, __LINE__))


__device__ double atomicAddFP64(double* address, double val)
{
	// https://devtalk.nvidia.com/default/topic/529341/speed-of-double-precision-cuda-atomic-operations-on-kepler-k20/
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}


__global__ void atomicAdd_test(unsigned int numInputs, float * d_inputData, int * d_accumulator, unsigned int * d_start, unsigned int * d_stop){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
	unsigned int start_time = 0;
	unsigned int stop_time = 0;

	if(tid < numInputs){
		start_time = clock();
		atomicAdd(d_accumulator, d_inputData[tid] * INTEGER_SCALE_FACTOR);
		stop_time = clock();

		d_start[tid] = start_time;
		d_stop[tid] = stop_time;
	}
}

__global__ void atomicAdd_test(unsigned int numInputs, float * d_inputData, unsigned int * d_accumulator, unsigned int * d_start, unsigned int * d_stop){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
	unsigned int start_time = 0;
	unsigned int stop_time = 0;

	if(tid < numInputs){
		start_time = clock();
		atomicAdd(d_accumulator, d_inputData[tid] * INTEGER_SCALE_FACTOR);
		stop_time = clock();

		d_start[tid] = start_time;
		d_stop[tid] = stop_time;
	}
}

__global__ void atomicAdd_test(unsigned int numInputs, float * d_inputData, float * d_accumulator, unsigned int * d_start, unsigned int * d_stop){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
	unsigned int start_time = 0;
	unsigned int stop_time = 0;

	if(tid < numInputs){
		start_time = clock();
		atomicAdd(d_accumulator, d_inputData[tid]);
		stop_time = clock();

		d_start[tid] = start_time;
		d_stop[tid] = stop_time;
	}
}

__global__ void atomicAdd_test(unsigned int numInputs, double * d_inputData, double * d_accumulator, unsigned int * d_start, unsigned int * d_stop){
	unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);
	unsigned int start_time = 0;
	unsigned int stop_time = 0;

	if(tid < numInputs){
		start_time = clock();
#if CUDA_VERSION >= 8000 && __CUDA_ARCH__ >= 600
		atomicAdd(d_accumulator, d_inputData[tid]);
#else 
		atomicAddFP64(d_accumulator, d_inputData[tid]);
#endif
		stop_time = clock();

		d_start[tid] = start_time;
		d_stop[tid] = stop_time;
	}
}

void generateInputData(unsigned int numInputs, unsigned long long int seed, float * d_data){

	hiprandGenerator_t rng = NULL;

	// Create RNG, seed RNG and populate device array.
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT); // @todo - curand error check
	hiprandSetPseudoRandomGeneratorSeed(rng, seed); // @todo - curand error check

	hiprandGenerateUniform(rng, d_data, numInputs); // @todo - curand error check
	
	// Cleanup rng
	hiprandDestroyGenerator(rng); // @todo - curand error check

}

void generateInputData(unsigned int numInputs, unsigned long long int seed, double * d_data){

	hiprandGenerator_t rng = NULL;

	// Create RNG, seed RNG and populate device array.
	hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT); // @todo - curand error check
	hiprandSetPseudoRandomGeneratorSeed(rng, seed); // @todo - curand error check

	hiprandGenerateUniformDouble(rng, d_data, numInputs); // @todo - curand error check
	
	// Cleanup rng
	hiprandDestroyGenerator(rng); // @todo - curand error check

}


void printAccumulatorTotal(int v){
	fprintf(stdout, "Accumulator: %d\n", v);
	fflush(stdout);
}
void printAccumulatorTotal(long long int v){
	fprintf(stdout, "Accumulator: %ll\n", v);
	fflush(stdout);
}
void printAccumulatorTotal(unsigned int v){
	fprintf(stdout, "Accumulator: %u\n", v);
	fflush(stdout);
}
void printAccumulatorTotal(unsigned long long int v){
	fprintf(stdout, "Accumulator: %llu\n", v);
	fflush(stdout);
}
void printAccumulatorTotal(float v){
	fprintf(stdout, "Accumulator: %f\n", v);
	fflush(stdout);
}
void printAccumulatorTotal(double v){
	fprintf(stdout, "Accumulator: %f\n", v);
	fflush(stdout);
}

template <typename T, typename U>
void runAtomicAddTest(unsigned int numIterations, unsigned int numInputs, unsigned long long int seed){

	T *h_accumulator = (T*)malloc(1 * sizeof(T));
	U *h_inputData = (U*)malloc(numInputs * sizeof(U)); //@todo - unneeded?

	T *d_accumulator = NULL;
	U *d_inputData = NULL;

	fprintf(stdout, "atomicAdd(%s) RNG(%s) %d threads %d iterations seed %d\n", typeid(*h_accumulator).name(), typeid(*h_inputData).name(), numInputs, numIterations, seed);
	fflush(stdout);

	unsigned int *h_start = (unsigned int*)malloc(numInputs * sizeof(unsigned int));
	unsigned int *h_stop = (unsigned int*)malloc(numInputs * sizeof(unsigned int));
	
	unsigned int *d_start = NULL;
	unsigned int *d_stop = NULL;

	// Create cudaEvents for timing.
	hipEvent_t start, stop;
	float milliseconds = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate device data.
	CUDA_CALL(hipMalloc((void**)&d_accumulator, 1 * sizeof(T)));
	CUDA_CALL(hipMalloc((void**)&d_inputData, numInputs * sizeof(U)));
	CUDA_CALL(hipMalloc((void**)&d_start, numInputs * sizeof(unsigned int)));
	CUDA_CALL(hipMalloc((void**)&d_stop, numInputs * sizeof(unsigned int)));


	// Initialise accumulator
	(*h_accumulator) = (T)0.0;
	CUDA_CALL(hipMemcpy(d_accumulator, h_accumulator, 1 * sizeof(T), hipMemcpyHostToDevice));

	// Generate random data
	generateInputData(numInputs, seed, d_inputData);

	// Get a function pointer to the kernel for this data type.
	void (*kernel)(unsigned int, U*, T*, unsigned int*, unsigned int*) = atomicAdd_test;

	// Accumulate values via kernel.
	int blockSize, minGridSize, gridSize;
	CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel, 0, numInputs));
	gridSize = (numInputs + blockSize - 1) / blockSize;

	CUDA_CALL(hipEventRecord(start));
	kernel<<<gridSize, blockSize>>>(numInputs, d_inputData, d_accumulator, d_start, d_stop);
	CUDA_CHECK();
	CUDA_CALL(hipEventRecord(stop));

	// Copy Data from Device to Host
	CUDA_CALL(hipMemcpy(h_accumulator, d_accumulator, 1 * sizeof(T), hipMemcpyDeviceToHost));
	//CUDA_CALL(cudaMemcpy(h_inputData, d_inputData, numInputs * sizeof(U), cudaMemcpyDeviceToHost)); //@todo - remove?
	CUDA_CALL(hipMemcpy(h_start, d_start, numInputs * sizeof(unsigned int), hipMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(h_stop, d_stop, numInputs * sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	CUDA_CALL(hipEventSynchronize(stop));
	CUDA_CALL(hipEventElapsedTime(&milliseconds, start, stop));

#if defined(VERBOSE) && VERBOSE > 0

	// Find the minimum start time.
	unsigned int min = h_start[0];
	for(unsigned int i = 0; i < numInputs; i++){
		min = (h_start[i] < min) ? h_start[i] : min;
	}

	fprintf(stdout, "thread, warp, start, stop\n");
	for(unsigned int i = 0; i < numInputs; i++){
		if (i > 0 && i % 32 == 0){
			fprintf(stdout, "---------------\n");
		}
		fprintf(stdout, "%d, %d, %u, %u\n", i % 32, i / 32, h_start[i] - min, h_stop[i] - min); 
	}
	fflush(stdout);
#endif 

	// Print output messages
	printAccumulatorTotal(h_accumulator[0]);
	fprintf(stdout, "Time: %f milliseconds\n");

	// Destroy events
	CUDA_CALL(hipEventDestroy(start));
	CUDA_CALL(hipEventDestroy(stop));
	// Free device data
	CUDA_CALL(hipFree(d_accumulator));
	CUDA_CALL(hipFree(d_inputData));
	CUDA_CALL(hipFree(d_start));
	CUDA_CALL(hipFree(d_stop));

	// Free host data
	free(h_accumulator);
	free(h_inputData);
	free(h_start);
	free(h_stop);

	// Reset the device for profiler output.
	CUDA_CALL(hipDeviceReset());

	fprintf(stdout, "\n");
	fflush(stdout);
}


void checkUsage(
	int argc,
	char *argv[],
	unsigned int *numIterations,
	unsigned int *numElements,
	unsigned long long int *seed
	){
		// If an incorrect number of arguments is specified, print usage.
		if (argc < MIN_ARGS || argc > MAX_ARGS){
			const char *usage = "Usage: \n"
				"%s <num_iterations> <num_elements> <seed>\n"
				"\n"
				"    <num_iterations> number of atomicAdd iterations to repeat (default %u)\n"
				"    <num_elements>   number of threads to launch (default %u)\n"
				"    <seed>           seed for RNG (default %llu)\n"
				"\n";
			fprintf(stdout, usage, EXE_NAME, DEFAULT_NUM_ITERATIONS, DEFAULT_NUM_ELEMENTS, DEFAULT_SEED);
			fflush(stdout);
			exit(EXIT_FAILURE);
		}

		// If there are more than 1 arg (the filename)5
		if(argc > MIN_ARGS){
			// Extract the number of iterations
			(*numIterations) = (unsigned int) atoi(argv[ARG_ITERATIONS]);
			// Extract the number of elements
			(*numElements) = (unsigned int) atoi(argv[ARG_ELEMENTS]);
			// Extract the seed
			(*seed) = strtoull(argv[ARG_SEED], nullptr, 0);
		}

#if defined(VERBOSE) && VERBOSE > 0
		printf("iterations: %u\n", numIterations);
		printf("threads:    %u\n", numElements);
		printf("seed:       %llu\n", seed);
#endif

}

int main(int argc, char *argv[])
{
	unsigned int numIterations = DEFAULT_NUM_ITERATIONS;
	unsigned int numElements = DEFAULT_NUM_ELEMENTS;
	unsigned long long int seed = DEFAULT_SEED;

	checkUsage(argc, argv, &numIterations, &numElements, &seed);

	runAtomicAddTest<float, float>(numIterations, numElements, seed);
	//runAtomicAddTest<double, double>(numIterations, numElements, seed);
	//runAtomicAddTest<int, float>(numIterations, numElements, seed);
	//runAtomicAddTest<long long int, float>(numIterations, numElements, seed);


    return 0;
}
